#include "hip/hip_runtime.h"
// Andrew Gloster
// May 2018
// Example of y direction periodic 2D code

//   Copyright 2018 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.


// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <cmath>
#include <iostream>
#include <cstdio>
#include "hip/hip_runtime.h"

// ---------------------------------------------------------------------
// cuSten - Note the file position is relative
// ---------------------------------------------------------------------

#include "../../cuSten/cuSten.h"

// ---------------------------------------------------------------------
// MACROS
// ---------------------------------------------------------------------

#define BLOCK_X 16
#define BLOCK_Y 16


// ---------------------------------------------------------------------
// Main Program
// ---------------------------------------------------------------------

int main()
{	
	// Set the device number
	int deviceNum = 0;

	// Declare Domain Size
	int nx = 512;
	int ny = 512;

	double ly = 2 * M_PI;

	// Domain spacings
	double dy = ly / (double) (nx);

	// Set the number of tiles per device
	int numTiles = 4;

	// Initial Conditions
	double* dataOld;
	double* dataNew;
	double* answer;

	// -----------------------------
	// Allocate the memory 
	// -----------------------------

	hipMallocManaged(&dataOld, nx * ny * sizeof(double));
	hipMallocManaged(&dataNew, nx * ny * sizeof(double));
	hipMallocManaged(&answer, nx * ny * sizeof(double));

	// -----------------------------
	// Set the initial conditions
	// -----------------------------

	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			dataOld[j * nx + i] = sin(j * dy);
			dataNew[j * nx + i] = 0.0;
			answer[j * nx + i] =- sin(j * dy);
		}
	}


	// // Ensure all the above is completed
	hipDeviceSynchronize();

	// -----------------------------
	// Set the stencil to compute
	// -----------------------------

	int numSten = 3;
	int numStenTop = 1;
	int numStenBottom = 1;

	double* weights;
	hipMallocManaged(&weights, numSten * sizeof(double));

	weights[0] = 1.0 / pow(dy, 2.0);
	weights[1] = - 2.0 / pow(dy, 2.0);
	weights[2] = 1.0 / pow(dy, 2.0);

	// -----------------------------
	// Set up device
	// -----------------------------

	// Number of points per device, subdividing in y
	int nxDevice = nx;
	int nyDevice = ny;

	// Set up the compute device structs
	cuSten_t yDirCompute;

	// Initialise the instance of the stencil
	custenCreate2DYp(&yDirCompute, deviceNum, numTiles, nxDevice, nyDevice, BLOCK_X, BLOCK_Y, dataNew, dataOld, weights, numSten, numStenTop, numStenBottom);

	// Synchronise to ensure everything initialised
	hipDeviceSynchronize();

	// -----------------------------
	// Compute
	// -----------------------------

	// Run the computation
	custenCompute2DYp(&yDirCompute, 0);

	// // Synchronise at the end to ensure everything is complete
	hipDeviceSynchronize();

	for (int j = 0; j < 256; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			printf("%lf %lf %d %d \n", dataNew[j * nx + i], answer[j * nx + i], i, j);
		}
	}

	// -----------------------------
	// Destroy struct and free memory
	// -----------------------------

	// Destroy struct
	custenDestroy2DYp(&yDirCompute);

	// Free memory at the end
	hipFree(dataOld);
	hipFree(dataNew);
	hipFree(answer);
	hipFree(weights);
	
	// Return 0 when the program completes
	return 0;
}