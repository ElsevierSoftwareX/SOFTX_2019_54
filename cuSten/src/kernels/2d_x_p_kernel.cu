#include "hip/hip_runtime.h"
// Andrew Gloster
// May 2018
// Kernel to apply an x direction stencil on a 2D grid - non periodic

//   Copyright 2018 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.


// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <iostream>
#include <cstdio>

// ---------------------------------------------------------------------
// Custom libraries and headers
// ---------------------------------------------------------------------

#include "../util/util.h"
#include "../struct/cuSten_struct_type.h"

// ---------------------------------------------------------------------
//  Kernel Definition
// ---------------------------------------------------------------------

__global__ void kernel2DXp
(

	double* dataOutput,  					// Answer data

	double* dataInput,					// Input data

	const double* d_weights,       		// Stencil weights

	const int numSten,					// Stencil width
	const int numStenLeft,				// Number of points to the left
	const int numStenRight,				// Number of points to the right

	const int nxLocal,					// Number of points in shared memory in x direction
	const int nyLocal,					// Number of points in shared memory in y direction

	const int BLOCK_X,					// Number of threads in block in y

	const int nxDevice					// Number of points in x on the device
)
{	
	// -----------------------------	
	// Allocate the shared memory
	// -----------------------------

	extern __shared__ int memory[];
	
	double* arrayLocal = (double*)&memory;
	double* weigthsLocal = (double*)&arrayLocal[nxLocal * nyLocal];

	// Move the weigths into shared memory
	#pragma unroll
	for (int k = 0; k < numSten; k++)
	{
		weigthsLocal[k] = d_weights[k];
	}

	// -----------------------------
	// Set the indexing
	// -----------------------------

	// True matrix index
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

	// Local matrix index
	int localIdx = threadIdx.x + numStenLeft;
	int localIdy = threadIdx.y;

	// Local sum variable
	double sum = 0.0;

	// Set index for summing stencil
	int stenSet;

	// -----------------------------
	// Set interior
	// -----------------------------

	arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nxDevice + globalIdx];

	// -----------------------------
	// Set x boundaries
	// -----------------------------

	// If block is in the interior
	if (blockIdx.x != 0 && blockIdx.x != nxDevice / BLOCK_X - 1)
	{

		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nxDevice + (globalIdx - numStenLeft)];
		}

		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nxDevice + globalIdx + BLOCK_X];
		}
	}

	// If block is on the left boundary
	if (blockIdx.x == 0)
	{
		arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nxDevice + globalIdx];

		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nxDevice + (nxDevice - numStenLeft + threadIdx.x)];
		}

		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nxDevice + globalIdx + BLOCK_X];
		}
	}

	// Set the right boundary blocks
	if (blockIdx.x == nxDevice / BLOCK_X - 1)
	{
		arrayLocal[localIdy * nxLocal + threadIdx.x + numStenLeft] = dataInput[globalIdy * nxDevice + globalIdx];

		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nxDevice + (globalIdx - numStenLeft)];
		}

		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nxDevice + threadIdx.x];
		}
	}

	// -----------------------------
	// Compute the stencil
	// -----------------------------

	__syncthreads();

	stenSet = localIdy * nxLocal + threadIdx.x;

	#pragma unroll
	for (int k = 0; k < numSten; k++)
	{
		sum += weigthsLocal[k] * arrayLocal[stenSet + k];
	}

	__syncthreads();

	// -----------------------------
	// Copy back to global
	// -----------------------------

	dataOutput[globalIdy * nxDevice + globalIdx] = sum;
}

// ---------------------------------------------------------------------
// Function to compute kernel
// ---------------------------------------------------------------------

void custenCompute2DXp
(
	cuSten_t* pt_cuSten,

	bool offload
)
{	
	// Buffer used for error checking
	char msgStringBuffer[1024];

	// Set current active compute device
	hipSetDevice(pt_cuSten->deviceNum);
	sprintf(msgStringBuffer, "Setting current device to GPU %d", pt_cuSten->deviceNum);
	checkError(msgStringBuffer);

	dim3 blockDim(pt_cuSten->BLOCK_X, pt_cuSten->BLOCK_Y);
	dim3 gridDim(pt_cuSten->xGrid, pt_cuSten->yGrid);

	// Local memory grid sizes
	int local_nx = pt_cuSten->BLOCK_X + pt_cuSten->numStenLeft + pt_cuSten->numStenRight;
	int local_ny = pt_cuSten->BLOCK_Y;

	// Load the weights
	hipMemPrefetchAsync(pt_cuSten->weights, pt_cuSten->numSten * sizeof(double), pt_cuSten->deviceNum, pt_cuSten->streams[1]);

	// Preload the first block
	hipStreamSynchronize(pt_cuSten->streams[1]);
	hipMemPrefetchAsync(pt_cuSten->dataInput[0], pt_cuSten->nxDevice * pt_cuSten->nyTile * sizeof(double), pt_cuSten->deviceNum, pt_cuSten->streams[1]);
	hipMemPrefetchAsync(pt_cuSten->dataOutput[0], pt_cuSten->nxDevice * pt_cuSten->nyTile * sizeof(double), pt_cuSten->deviceNum, pt_cuSten->streams[1]);
	hipEventRecord(pt_cuSten->events[1], pt_cuSten->streams[1]);

	// Temporary stream and event used for permuting
	hipStream_t ts;
	hipEvent_t te;

	// Loop over the tiles
	for (int tile = 0; tile < pt_cuSten->numTiles; tile++)
	{
		// Synchronise the events to ensure computation overlaps
		hipEventSynchronize(pt_cuSten->events[0]);
		hipEventSynchronize(pt_cuSten->events[1]);

		// Preform the computation on the current tile
		kernel2DXp<<<gridDim, blockDim, pt_cuSten->mem_shared, pt_cuSten->streams[0]>>>(pt_cuSten->dataOutput[tile], pt_cuSten->dataInput[tile], pt_cuSten->weights, pt_cuSten->numSten, pt_cuSten->numStenLeft, pt_cuSten->numStenRight, local_nx, local_ny, pt_cuSten->BLOCK_X, pt_cuSten->nxDevice);
		hipEventRecord(pt_cuSten->events[0], pt_cuSten->streams[0]);

		// Offload should the user want to
		if (offload == 1)
		{
			hipMemPrefetchAsync(pt_cuSten->dataOutput[tile], pt_cuSten->nxDevice * pt_cuSten->nyTile * sizeof(double), hipCpuDeviceId, pt_cuSten->streams[0]);
	    	hipMemPrefetchAsync(pt_cuSten->dataInput[tile], pt_cuSten->nxDevice * pt_cuSten->nyTile * sizeof(double), hipCpuDeviceId, pt_cuSten->streams[0]);
		}

		// Load the next tile
    	if (tile < pt_cuSten->numTiles - 1)
    	{
    		hipStreamSynchronize(pt_cuSten->streams[1]);
			hipMemPrefetchAsync(pt_cuSten->dataOutput[tile + 1], pt_cuSten->nxDevice * pt_cuSten->nyTile * sizeof(double), pt_cuSten->deviceNum, pt_cuSten->streams[1]);
		 	hipMemPrefetchAsync(pt_cuSten->dataInput[tile + 1], pt_cuSten->nxDevice * pt_cuSten->nyTile * sizeof(double), pt_cuSten->deviceNum, pt_cuSten->streams[1]);
			hipEventRecord(pt_cuSten->events[1], pt_cuSten->streams[1]);
    	}

    	// Permute streams
    	for (int i = 0; i < pt_cuSten->numStreams - 1; i++)
    	{
    		ts = pt_cuSten->streams[i];
    		pt_cuSten->streams[i] = pt_cuSten->streams[i + 1];	
    		pt_cuSten->streams[i + 1] = ts;    			
    	}

    	// Permute events
		te = pt_cuSten->events[0]; pt_cuSten->events[0] = pt_cuSten->events[1]; pt_cuSten->events[1] = te; 
    }
}

// ---------------------------------------------------------------------
// End of file
// ---------------------------------------------------------------------