// Andrew Gloster
// January 2019
// Functions to create and destroy the required struct for a 2D y direction
// non periodic calculation

//   Copyright 2019 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.


// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <iostream>

// ---------------------------------------------------------------------
// Custom libraries and headers
// ---------------------------------------------------------------------

#include "cuSten_struct_type.h"
#include "cuSten_struct_functions.h"
#include "../util/util.h"

// ---------------------------------------------------------------------
// Function to create the struct for a 2D y direction non periodic user function
// ---------------------------------------------------------------------

void custenCreate2DYnpFun(
	cuSten_t* pt_cuSten,		// Pointer to the compute struct type

	int deviceNum,				// Device on which to compute this stencil

	int numTiles,				// Number of tiles to divide the data on the device into

	int nxDevice,				// Number of points in x on the device
	int nyDevice,				// Number of points in y on the device

	int BLOCK_X,				// Number of threads to use in x
	int BLOCK_Y,				// Number of threads to use in y

	double* dataNew,			// Output data
	double* dataOld,			// Input data
	double* coe,				// Arracy containing the weights

	int numSten,				// Number of points in a stencil
	int numStenTop,				// Number of points in the top of the stencil
	int numStenBottom,			// Number of points in the bottom of the stencil

	double* func 				// User defined function
) 
{
	// Buffer used for error checking
	char msgStringBuffer[1024];

	// Set the device number associated with the struct
  	pt_cuSten->deviceNum = deviceNum;

  	// Set the number of streams
  	pt_cuSten->numStreams = 3;

  	// Set the number of tiles
  	pt_cuSten->numTiles = numTiles;

  	// Set the number points in x on the device
  	pt_cuSten->nxDevice = nxDevice;

  	// Set the number points in y on the device
  	pt_cuSten->nyDevice = nyDevice;

  	// Number of threads in x on the device
	pt_cuSten->BLOCK_X = BLOCK_X;

  	// Number of threads in y on the device
	pt_cuSten->BLOCK_Y = BLOCK_Y;

	// Set current active compute device
	hipSetDevice(pt_cuSten->deviceNum);
	sprintf(msgStringBuffer, "Setting current device to GPU %d", pt_cuSten->deviceNum);
	checkError(msgStringBuffer);	

	// Create memeory for the streams
	pt_cuSten->streams = (hipStream_t*)malloc(pt_cuSten->numStreams * sizeof(hipStream_t*));

	// Create the streams
	for (int st = 0; st < pt_cuSten->numStreams; st++)
	{
		hipStreamCreate(&pt_cuSten->streams[st]);
		sprintf(msgStringBuffer, "Creating stream %d on GPU %d", st, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);	
	}

	// Create memeory for the events
	pt_cuSten->events = (hipEvent_t*)malloc(2 * sizeof(hipEvent_t*));

	// Create the events
	for (int ev = 0; ev < 2; ev++)
	{
		hipEventCreate(&pt_cuSten->events[ev]);
		sprintf(msgStringBuffer, "Creating event %d on GPU %d", ev, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);
	}

	// Set number of points in the stencil
	pt_cuSten->numSten = numSten;

	// Set number of points to the left in the stencil
	pt_cuSten->numStenTop = numStenTop;

	// Set number of points to the right in the stencil
	pt_cuSten->numStenBottom = numStenBottom;

	// Set local block array sizes - x direction
	pt_cuSten->nxLocal = pt_cuSten->BLOCK_X;

	// Set loacl block array sizes - y direction
	pt_cuSten->nyLocal = pt_cuSten->BLOCK_Y + pt_cuSten->numStenTop + pt_cuSten->numStenBottom;

	// Set the amount of shared memory required
	pt_cuSten->mem_shared = (pt_cuSten->nxLocal * pt_cuSten->nyLocal) * sizeof(double) + pt_cuSten->numSten * sizeof(double);

	// Find number of points per tile
	pt_cuSten->nyTile = pt_cuSten->nyDevice / pt_cuSten->numTiles;	

	// Set the grid up
    pt_cuSten->xGrid = (pt_cuSten->nxDevice % pt_cuSten->BLOCK_X == 0) ? (pt_cuSten->nxDevice / pt_cuSten->BLOCK_X) : (pt_cuSten->nxDevice / pt_cuSten->BLOCK_X + 1);
    pt_cuSten->yGrid = (pt_cuSten->nyTile % pt_cuSten->BLOCK_Y == 0) ? (pt_cuSten->nyTile / pt_cuSten->BLOCK_Y) : (pt_cuSten->nyTile / pt_cuSten->BLOCK_Y + 1);

	// Set the device weights pointer
	pt_cuSten->coe = coe;

	// Allocate the pointers for each input tile
	pt_cuSten->dataInput = (double**)malloc(pt_cuSten->numTiles * sizeof(double));

	// Allocate the pointers for each output tile
	pt_cuSten->dataOutput = (double**)malloc(pt_cuSten->numTiles * sizeof(double));

	// // Tile offset index
	int offset = pt_cuSten->nxDevice * pt_cuSten->nyTile;

	// // Match the pointers to the data
	for (int tile = 0; tile < pt_cuSten->numTiles; tile++)
	{	
		// Set the input data
		pt_cuSten->dataInput[tile] = &dataOld[tile * offset];

		// Set the output data
		pt_cuSten->dataOutput[tile] = &dataNew[tile * offset];
	}

	// Create cases depending on what tile numbers - Periodic
	// 1 tile
	// 2 tiles
	// 3 or greater

	// Allocate top boundary memory
	pt_cuSten->boundaryTop = (double**)malloc(pt_cuSten->numTiles * sizeof(double));

	// Allocate bottom boundary memory
	pt_cuSten->boundaryBottom = (double**)malloc(pt_cuSten->numTiles * sizeof(double));

	switch(pt_cuSten->numTiles)
	{
		// One tile only requires single top and bottom to be set
		case 1:
			pt_cuSten->boundaryTop[0] = &dataOld[0];
			pt_cuSten->boundaryBottom[0] = &dataOld[(pt_cuSten->nyDevice - pt_cuSten->numStenBottom) * pt_cuSten->nxDevice]; 

			break;

		// Two tiles requires a special case of only setting two tiles
		case 2:
			pt_cuSten->boundaryTop[0] = &dataOld[0];
			pt_cuSten->boundaryBottom[0] = &dataOld[pt_cuSten->nyTile * pt_cuSten->nxDevice];

			pt_cuSten->boundaryTop[1] = &dataOld[(pt_cuSten->nyTile - pt_cuSten->numStenTop) * pt_cuSten->nxDevice];
			pt_cuSten->boundaryBottom[1] = &dataOld[(pt_cuSten->nyDevice - pt_cuSten->numStenBottom) * pt_cuSten->nxDevice];

			break;

		// Default case has interiors, so set the top tile, then loop over interior, then set the bottom tile
		default:
			pt_cuSten->boundaryTop[0] = &dataOld[0];
			pt_cuSten->boundaryBottom[0] = &dataOld[pt_cuSten->nyTile * pt_cuSten->nxDevice];

			for (int tile = 1; tile < pt_cuSten->numTiles - 1; tile++)
			{
				pt_cuSten->boundaryTop[tile] = &dataOld[(pt_cuSten->nyTile * tile - pt_cuSten->numStenTop) * pt_cuSten->nxDevice];
				pt_cuSten->boundaryBottom[tile] = &dataOld[(pt_cuSten->nyTile * (tile + 1)) * pt_cuSten->nxDevice];
			}

			pt_cuSten->boundaryTop[pt_cuSten->numTiles - 1] = &dataOld[(pt_cuSten->nyTile * (pt_cuSten->numTiles - 1) - pt_cuSten->numStenTop) * pt_cuSten->nxDevice];
			pt_cuSten->boundaryBottom[pt_cuSten->numTiles - 1] = &dataOld[(pt_cuSten->nyDevice - pt_cuSten->numStenBottom) * pt_cuSten->nxDevice];

			break;
	}

	// Number of points in top boundary data
	pt_cuSten->numBoundaryTop = pt_cuSten->numStenTop * pt_cuSten->nxDevice;

	// Number of points in bottom boundary data
	pt_cuSten->numBoundaryBottom = pt_cuSten->numStenBottom * pt_cuSten->nxDevice;

	// Set the function
	pt_cuSten->devFunc = func;

}

// ---------------------------------------------------------------------
// Function to destroy the struct for a 2D y direction non periodic user function
// ---------------------------------------------------------------------

void custenDestroy2DYnpFun(
	cuSten_t* pt_cuSten
) 
{
	// Buffer used for error checking
	char msgStringBuffer[1024];

	// Set current active compute device
	hipSetDevice(pt_cuSten->deviceNum);
	sprintf(msgStringBuffer, "Setting current device to GPU %d", pt_cuSten->deviceNum);
	checkError(msgStringBuffer);	


	// Destroy the streams
	for (int st = 0; st < pt_cuSten->numStreams; st++)
	{
		hipStreamDestroy(pt_cuSten->streams[st]);
		sprintf(msgStringBuffer, "Destroying stream %d on GPU %d", st, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);	
	}

	// Free the main memory
	free(pt_cuSten->streams);

	// // Create the events
	for (int ev = 0; ev < 2; ev++)
	{
		hipEventDestroy(pt_cuSten->events[ev]);
		sprintf(msgStringBuffer, "Destroying event %d on GPU %d", ev, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);
	}

	// Free the main memory
	free(pt_cuSten->events);

	// Free the pointers for each input tile
	free(pt_cuSten->dataInput);

	// Free the pointers for each output tile
	free(pt_cuSten->dataOutput);

	// Free the top boundary tile pointers
	free(pt_cuSten->boundaryTop);

	// Free the bottom boundary tile pointers
	free(pt_cuSten->boundaryBottom);
}


// ---------------------------------------------------------------------
// End of file
// ---------------------------------------------------------------------