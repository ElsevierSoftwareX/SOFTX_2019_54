#include "hip/hip_runtime.h"
// Andrew Gloster
// May 2018

//   Copyright 2018 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.

/*! \file error.cu
// Functions to catch errors in the cuSten library
*/

// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <iostream>

// ---------------------------------------------------------------------
// Custom libraries and headers
// ---------------------------------------------------------------------

#include "util.h"

// ---------------------------------------------------------------------
//  Error checking function
// ---------------------------------------------------------------------

/*! \fun __global__ void checkError
	\brief Function responds to cudaErrors thrown
    \param action Pointer to string input by programmer
*/

void checkError(const char* action) 
{
  
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_FAILURE);
  }
}

// ---------------------------------------------------------------------
// End of file
// ---------------------------------------------------------------------