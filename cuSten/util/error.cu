// Andrew Gloster
// May 2018
// Functions to catch errors in the cuSten library

// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <iostream>

// ---------------------------------------------------------------------
// Custom libraries and headers
// ---------------------------------------------------------------------

#include "util.h"

// ---------------------------------------------------------------------
//  Error checking function
// ---------------------------------------------------------------------

void checkError (const char* action) 
{
  
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_FAILURE);
  }
}

// ---------------------------------------------------------------------
// End of file
// ---------------------------------------------------------------------