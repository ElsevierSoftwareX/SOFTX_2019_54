#include "hip/hip_runtime.h"
// Andrew Gloster
// February 2019
// Program to solve the 2D Cahn-Hilliard equation on a periodic domain using the ADI method
// Outputs timing

//   Copyright 2019 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.

// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "hdf5.h"
#include <time.h>

// ---------------------------------------------------------------------
//  Programmer Libraries and Headers
// ---------------------------------------------------------------------

#include "../../../cuSten/cuSten.h"
#include "cuPentBatch.h"
#include "BatchHyper.h"

// ---------------------------------------------------------------------
// MACROS
// ---------------------------------------------------------------------

// Block sizes for finding RHS
#define BLOCK_X_FUN 8
#define BLOCK_Y_FUN 8

#define BLOCK_X 32
#define BLOCK_Y 32

// Block size for inverting
#define BLOCK_INV 64

//---------------------------------------------------------------------
// Static functions for use in main program
//---------------------------------------------------------------------

// Find cBar for differencing
__global__ static void findCBar(double* cOld, double* cCurr, double* cBar, int nx)
{
	// Matrix index
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

	// Set index being computed
	int index = globalIdy * nx + globalIdx;

	// Find cBar
	cBar[index] = 2.0 * cCurr[index] - cOld[index];
}

// Find the full combined RHS
__global__ static void findRHS(double* cOld, double* cCurr, double* cHalf, double* cNonLinRHS, int nx)
{
	// Matrix index
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

	// Set index being computed
	int index = globalIdy * nx + globalIdx;

	// Set the RHS for inversion
	cHalf[index] += - (2.0 / 3.0) * (cCurr[index] - cOld[index]) + cNonLinRHS[index];

	// Set cOld to cCurr
	cOld[index] = cCurr[index];
}

// Recover the updated timestep
__global__ static void findNew(double* cCurr, double* cBar, double* cHalf, int nx)
{
	// Matrix index
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

	// Set index being computed
	int index = globalIdy * nx + globalIdx;

	// Recover the new data
	cCurr[index] = cBar[index] + cHalf[index];
}

static double double_rand(double min, double max)
{
    double scale = (double) rand() / (double) RAND_MAX; /* [0, 1.0] */
    return min + scale * ( max - min );      /* [min, max] */
}

//---------------------------------------------------------------------
// Function to calculate the non linear RHS
//---------------------------------------------------------------------

/*! \var typedef double (*devArg1X)(double*, double*, int);
    \brief The function pointer containing the user defined function to be applied <br>
    Input 1: The pointer to input data to the function <br>
    Input 2: The pointer to the coefficients provided by the user <br>
    Input 3: The current index position (centre of the stencil to be applied) <br>
    Input 4: Value to be used to jump between rows. (j + 1, j - 1 etc.) <br>
    Input 5: Size of stencil in x direction <br>
    Input 6: Size of stencil in y direction
*/

typedef double (*devArg1XY)(double*, double*, int, int, int, int);

__inline__ __device__ double nonLinRHS(double* data, double* coe, int loc, int jump, int nx, int ny)
{	
	double result = 0.0;
	double current;
	int temp;
	int count = 0;

	#pragma unroll
	for (int j = 0; j < ny; j++)
	{
		temp = loc + j * jump;

		#pragma unroll
		for (int i = 0; i < nx; i++)
		{
			current = data[temp + i];

			result += coe[count] * ((current * current * current) - current);

			count ++;
		}
	}

	return result;
}

__device__ devArg1XY devFunc = nonLinRHS;


// ---------------------------------------------------------------------
//  Begin main program
// ---------------------------------------------------------------------

int main(int argc, char *argv[])
{
    //----------------------------------------
    // Simulation paramters
    //----------------------------------------

    // Set coefficients
    double D = 1.0;
    double gamma = 0.01;

    // Set grid spacing -- Use a square grid -- thus all n = ny
    // Read from command line
    int nx;
    nx = atoi(argv[1]);

    // Set the size of the reduced matrix
    int size = nx - 2;

    // Set timing
    double T = 10.0;

    // Domain size
    double lx = 16.0 * M_PI;

    // Spacings
    double dx = lx / nx;
    double dt = 0.1 * dx;

    //  Buffer used for error checking
    char msgStringBuffer[1024];

    // What device to compute on
    int computeDevice = 0;

    //----------------------------------------
    // Set up GPU grids
    //----------------------------------------

    // Set for inversion
    int gridInv = (nx % BLOCK_INV == 0) ? (nx / BLOCK_INV) : (nx / BLOCK_INV + 1);

    dim3 blockDimInv(BLOCK_INV);
    dim3 gridDimInv(gridInv);

    // Set for any standard grid
    int xGrid = (nx % BLOCK_X == 0) ? (nx / BLOCK_X) : (nx / BLOCK_X + 1);
    int yGrid = (nx % BLOCK_Y == 0) ? (nx / BLOCK_Y) : (nx / BLOCK_Y + 1);

    dim3 blockDim(BLOCK_X, BLOCK_Y);
    dim3 gridDim(xGrid, yGrid);

    //----------------------------------------
    // Memory allocation
    //----------------------------------------    
    
    // Old timestep
    double* cOld;
    hipMallocManaged(&cOld, nx * nx * sizeof(double));
   
    sprintf(msgStringBuffer, "Failed to allocate memory for cOld");
    checkError(msgStringBuffer);    

    // Current timestep
    double* cCurr;
    hipMallocManaged(&cCurr, nx * nx * sizeof(double));
   
    sprintf(msgStringBuffer, "Failed to allocate memory for cCurr");
    checkError(msgStringBuffer);

    // New timestep
    double* cNonLinRHS;
    hipMallocManaged(&cNonLinRHS, nx * nx * sizeof(double));
   
    sprintf(msgStringBuffer, "Failed to allocate memory for cNonLinRHS");
    checkError(msgStringBuffer);   

    // Intermediate step
    double* cBar;
    hipMallocManaged(&cBar, nx * nx * sizeof(double));
   
    sprintf(msgStringBuffer, "Failed to allocate memory for cBar");
    checkError(msgStringBuffer);  
    
    // Intermediate step
    double* cHalf;
    hipMallocManaged(&cHalf, nx * nx * sizeof(double));
    
    sprintf(msgStringBuffer, "Failed to allocate memory for cBar");
    checkError(msgStringBuffer); 

    //----------------------------------------
    // Initial Condition
    //---------------------------------------- 

    // Indexing
    int temp, index;

    for (int j = 0; j < nx; j++)
    {
        temp = j * nx;
        for (int i = 0; i < nx; i++)
        {
            index = temp + i;

            cOld[index] = double_rand(- 0.1, 0.1);
            cCurr[index] = cOld[index];
        }
    }

    //----------------------------------------
    // Allocate the memory for the LHS
    //----------------------------------------

    // Lowest diagonal
    double* ds;
    hipMallocManaged(&ds, size * nx * sizeof(double));

    sprintf(msgStringBuffer, "Failed to allocate memory for ds");
    checkError(msgStringBuffer);   

    // Lower diagonal
    double* dl;
    hipMallocManaged(&dl, size * nx * sizeof(double));

    sprintf(msgStringBuffer, "Failed to allocate memory for dl");
    checkError(msgStringBuffer);  

    // Main daigonal
    double* diag;
    hipMallocManaged(&diag, size * nx * sizeof(double));

    sprintf(msgStringBuffer, "Failed to allocate memory for diag");
    checkError(msgStringBuffer); 

    // Upper diagonal
    double* du;
    hipMallocManaged(&du, size * nx * sizeof(double));

    sprintf(msgStringBuffer, "Failed to allocate memory for du");
    checkError(msgStringBuffer);   

    // Highest diagonal
    double* dw;
    hipMallocManaged(&dw, size * nx * sizeof(double));

    sprintf(msgStringBuffer, "Failed to allocate memory for dw");
    checkError(msgStringBuffer);   

    //----------------------------------------
    // Set up cuBLAS
    //---------------------------------------- 

	// Set a handle
	hipblasHandle_t handleBLAS;

	// Set a status
	hipblasStatus_t statusBLAS;

	// Create the handle
	statusBLAS = hipblasCreate(&handleBLAS);

	// Set constants
	const double alpha = 1.0;
	const double beta = 0.0;

    //----------------------------------------
    // Set coefficients
    //---------------------------------------- 	

	// Linear coefficient
    double simgaLin = 2.0 * dt * D * gamma / (3.0 * (pow(dx, 4.0)));

    // Set the diagonal elements
    double a = simgaLin;
    double b = - 4 * simgaLin;
    double c = 1 + 6 * simgaLin;
    double d = - 4 * simgaLin;
    double e = simgaLin;

    //----------------------------------------
    // Set the matrix
    //----------------------------------------

    // Set the LHS for inversion
    setMultiLHS<<<gridDim, blockDim>>>(ds, dl, diag, du, dw, a, b, c, d, e, size, nx);

    sprintf(msgStringBuffer, "Failed to set LHS matrix for initial timestep");
    checkError(msgStringBuffer);

	// Ensure matrix is set
	hipDeviceSynchronize();

    // Pre-factor the LHS
    pentFactorBatch<<<gridDimInv, blockDimInv>>>(ds, dl, diag, du, dw, size, nx);

    sprintf(msgStringBuffer, "Failed to pre factor LHS matrix for initial timestep");
    checkError(msgStringBuffer);

	// Ensure matrix is factorised
	hipDeviceSynchronize();

    //----------------------------------------
    // Find omega and set inverses
    //----------------------------------------

    double* omega = (double*)malloc(4 * sizeof(double));
	if (omega == NULL)
	{
		printf("%s \n", "Failed to malloc omega");
	}

	double* inv1Single = (double*)malloc(size * sizeof(double));
	if (inv1Single == NULL)
	{
		printf("%s \n", "Failed to malloc inv1Single");
	}

	double* inv2Single = (double*)malloc(size * sizeof(double));
	if (inv2Single == NULL)
	{
		printf("%s \n", "Failed to malloc inv2Single");
	}

    double* inv1Multi;
    hipMallocManaged(&inv1Multi, nx * size * sizeof(double));

    sprintf(msgStringBuffer, "Failed to allocate memory for inv1Multi");
    checkError(msgStringBuffer); 

    double* inv2Multi;
    hipMallocManaged(&inv2Multi, nx * size * sizeof(double));

    sprintf(msgStringBuffer, "Failed to allocate memory for inv2Multi");
    checkError(msgStringBuffer); 

    findOmega(omega, inv1Single, inv2Single, a, b, c, d, e, nx);

    for (int j = 0; j < size; j++)
    {	
    	temp = j * nx;

    	for (int i = 0; i < nx; i++)
    	{
    		index = temp + i;

    		inv1Multi[index] = inv1Single[j]; 
    		inv2Multi[index] = inv2Single[j];
    	}
    }

    //----------------------------------------
    // Set compute for linear RHS
    //----------------------------------------

    int linHoriz = 5;
	int linLeft = 2;
	int linRight = 2;

	int linVert = 5;
	int linTop = 2;
	int linBottom = 2;

	double* weightsLinRHS;
	hipMallocManaged(&weightsLinRHS, linHoriz * linVert * sizeof(double));

	weightsLinRHS[0] = 0.0; 						weightsLinRHS[1] = 0.0;								weightsLinRHS[2] = - 1.0 * simgaLin;					weightsLinRHS[3] = 0.0;								weightsLinRHS[4] = 0.0;					
	weightsLinRHS[5] = 0.0;							weightsLinRHS[6] = - 2.0 * simgaLin; 				weightsLinRHS[7] = 8.0 * simgaLin;						weightsLinRHS[8] = - 2.0 * simgaLin;				weightsLinRHS[9] = 0.0; 			
	weightsLinRHS[10] = - 1.0 * simgaLin; 			weightsLinRHS[11] = 8.0 * simgaLin;					weightsLinRHS[12] = - 20.0 * simgaLin;					weightsLinRHS[13] = 8.0 * simgaLin;					weightsLinRHS[14] = - 1.0 * simgaLin;					
	weightsLinRHS[15] = 0.0;						weightsLinRHS[16] = - 2.0 * simgaLin; 				weightsLinRHS[17] = 8.0 * simgaLin;						weightsLinRHS[18] = - 2.0 * simgaLin;				weightsLinRHS[19] = 0.0;
	weightsLinRHS[20] = 0.0; 						weightsLinRHS[21] = 0.0;							weightsLinRHS[22] = -1.0 * simgaLin;					weightsLinRHS[23] = 0.0;							weightsLinRHS[24] = 0.0;					

	// Set up the compute device structs
	cuSten_t linRHS;

	// Set the number of tiles
	int linInitTiles = 1;

	// Initialise the instance of the stencil
	cuStenCreate2DXYp(&linRHS, computeDevice, linInitTiles, nx, nx, BLOCK_X, BLOCK_Y, cHalf, cBar, weightsLinRHS, linHoriz, linLeft, linRight, linVert, linTop, linBottom);
	
	// Ensure compute type created
	hipDeviceSynchronize();

    //----------------------------------------
    // Set up computation of non-linear RHS 
    //----------------------------------------   

	// Set up the compute device structs
	cuSten_t nonLinCompute;

	// Synchronise to ensure everything initialised
	hipDeviceSynchronize();

	// Copy the function to device memory
	double* func;
	hipMemcpyFromSymbol(&func, HIP_SYMBOL(devFunc), sizeof(devArg1XY));	

	// Set new non linear coefficient
	double sigmaNonLin = (dt / 3.0) * D * (2.0 / pow(dx, 2.0));

	int numStenHoriz = 3;
	int numStenLeft = 1;
	int numStenRight = 1;

	int numStenVert = 3;
	int numStenTop = 1;
	int numStenBottom = 1;

	int nonLinTiles = 1;

	double* coe;
	hipMallocManaged(&coe, numStenHoriz * numStenVert * sizeof(double));

	coe[0] = 0.0;						coe[1] = 1.0 * sigmaNonLin;				coe[2] = 0.0;
	coe[3] = 1.0 * sigmaNonLin;			coe[4] = - 4.0 * sigmaNonLin;			coe[5] = 1.0 * sigmaNonLin;
	coe[6] = 0.0;						coe[7] = 1.0 * sigmaNonLin;				coe[8] = 0.0;

	// Initialise the instance of the stencil
	cuStenCreate2DXYpFun(&nonLinCompute, computeDevice, nonLinTiles, nx, nx, BLOCK_X_FUN, BLOCK_Y_FUN, cNonLinRHS, cCurr, coe, numStenHoriz, numStenLeft, numStenRight, numStenVert, numStenTop, numStenBottom, func);

	// Synchronise to ensure everything initialised
	hipDeviceSynchronize();

    //----------------------------------------
    // Begin timestepping
    //----------------------------------------

    // Track current time-step
    double t = 0.0;

    // Define events for timing
    hipEvent_t start, stop;
  
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Store time
    float time;

    // Start time
    hipEventRecord(start, 0 );

    while (t < T)
    {
    	// Set cBar
        findCBar<<<gridDim, blockDim>>>(cOld, cCurr, cBar, nx);

     	// Ensure compute type created
    	hipDeviceSynchronize();

        // Compute the non-linear RHS
        cuStenCompute2DXYpFun(&nonLinCompute, 0);

        // Compute the linear RHS
        cuStenCompute2DXYp(&linRHS, 0);

        // Ensure compute type created
        hipDeviceSynchronize();

    	// Find the full RHS and then set cOld to cCurrent
    	findRHS<<<gridDim, blockDim>>>(cOld, cCurr, cHalf, cNonLinRHS, nx);

     	// Ensure compute type created
    	hipDeviceSynchronize();

    	// Transpose the result
    	statusBLAS = hipblasDgeam(handleBLAS, HIPBLAS_OP_T, HIPBLAS_OP_T, nx, nx, &alpha, cHalf, nx, &beta, NULL, nx, cCurr, nx);

    	if (statusBLAS != HIPBLAS_STATUS_SUCCESS) {
    		printf("Unable to compute transpose \n");
    		return EXIT_FAILURE;
    	}

    	// Ensure transpose completed
    	hipDeviceSynchronize();

        // Invert the matrix
        cyclicInv(ds, dl, diag, du, dw, inv1Multi, inv2Multi, omega, cCurr, a, b, d, e, BLOCK_INV, BLOCK_X, BLOCK_Y, size, nx);

    	// Transpose the result
    	statusBLAS = hipblasDgeam(handleBLAS, HIPBLAS_OP_T, HIPBLAS_OP_T, nx, nx, &alpha, cCurr, nx, &beta, NULL, nx, cHalf, nx);

    	if (statusBLAS != HIPBLAS_STATUS_SUCCESS) {
    		printf("Unable to compute transpose \n");
    		return EXIT_FAILURE;
    	}

    	// Ensure transpose completed
    	hipDeviceSynchronize();

        // Invert the matrix
        cyclicInv(ds, dl, diag, du, dw, inv1Multi, inv2Multi, omega, cHalf, a, b, d, e, BLOCK_INV, BLOCK_X, BLOCK_Y, size, nx);

    	// Ensure computation completed
    	hipDeviceSynchronize();

    	findNew<<<gridDim, blockDim>>>(cCurr, cBar, cHalf, nx);

    	// Ensure computation completed
    	hipDeviceSynchronize();


        // Add on the next time
        t += dt;
    }

   	// Ensure computation completed
	hipDeviceSynchronize(); 

    // End time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    // Get elapsed time for kernel execution
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("%f \n", time / 1000);

    //----------------------------------------
    // Free memory at the end
    //----------------------------------------

    free(omega);
    free(inv1Single);
    free(inv2Single);

	cuStenDestroy2DXYp(&linRHS);
	cuStenDestroy2DXYpFun(&nonLinCompute);

    hipFree(inv1Multi);
    hipFree(inv2Multi);

    hipFree(cOld);
    hipFree(cNonLinRHS);
    hipFree(cBar);
    hipFree(cHalf);

    hipFree(ds);
    hipFree(dl);
    hipFree(diag);
    hipFree(du);
    hipFree(dw);
}

// ---------------------------------------------------------------------
//  End main program
// ---------------------------------------------------------------------


// ---------------------------------------------------------------------
//  End of File
// ---------------------------------------------------------------------
